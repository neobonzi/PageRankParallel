#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <map>
#include <hipsparse.h>

#include "GraphUtils.h"
#include "cusparse_graph_utils.h"
#include "Node.h"
#include "CSVReader.h"
#include "SNAPReader.h"

#include <fcntl.h>
#include <math.h>
#include <stdio.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#ifndef ITERATIONS
#define ITERATIONS 1000 
#endif

#ifndef D
#define D 0.95
#endif

#ifndef max
#define max(a,b) ((a > b) ? a : b)
#endif
#define CSV_DELIMITER ","

using namespace std;
using namespace CSVReader;
using namespace SNAPReader;

// Handles an error for cuda calls.
static void HandleError( hipError_t err,
    const char *file,
    int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
    exit( EXIT_FAILURE );
  }
}

static void check_cusparse_err(hipsparseStatus_t err, const char* file,
                               int line) {
  if (err != HIPSPARSE_STATUS_SUCCESS) {
    printf( "%s in %s at line %d\n", ( err ),
        file, line );
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#define HANDLE_CUSPARSE( err ) (check_cusparse_err( err, __FILE__, __LINE__ ))

void PageRankKernel(double *val, int *rowptr, int *colndx,
                    double *p, double *pr, int width, int nnz, double *ones) {
   hipsparseHandle_t handle;
   HANDLE_CUSPARSE(hipsparseCreate(&handle));
   hipsparseMatDescr_t descr;
   HANDLE_CUSPARSE(hipsparseCreateMatDescr(&descr));
   double multiplier = D;
   double constant = (1-D)/(double)width;

   HANDLE_CUSPARSE(hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                  width, width, nnz, &multiplier,
                  descr,
                  val,
                  rowptr, colndx,
                  p, &constant,
                  pr)); 
   hipDeviceSynchronize();
}

void PageRankOnDevice(CSRMatrix *matrix, double *prestige, double *ones) {
   double *dVal, *dPrestige, *dPrestigeResult, *dOnes;
   int *dRowPtr, *dColInd;
   const int w = matrix->width;
   const int nnz = matrix->nnz;
   // Allocate memory.
   // CSR format
   HANDLE_ERROR(hipMalloc(&dVal, nnz*sizeof(double)));
   HANDLE_ERROR(hipMemcpy(dVal, matrix->csrValA, nnz*sizeof(double),
                hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMalloc(&dColInd, nnz*sizeof(int)));
   HANDLE_ERROR(hipMemcpy(dColInd, matrix->csrColIndA, nnz*sizeof(int),
                hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMalloc(&dRowPtr, (w+1)*sizeof(int)));
   HANDLE_ERROR(hipMemcpy(dRowPtr, matrix->csrRowPtrA, (w+1)*sizeof(int),
                hipMemcpyHostToDevice));
   // Prestige
   HANDLE_ERROR(hipMalloc(&dPrestige, w*sizeof(double)));
   HANDLE_ERROR(hipMemcpy(dPrestige, prestige, w*sizeof(double),
                hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMalloc(&dOnes, w*sizeof(double)));
   HANDLE_ERROR(hipMemcpy(dOnes, ones, w*sizeof(double),
                hipMemcpyHostToDevice));
   // Prestige Result
   HANDLE_ERROR(hipMalloc(&dPrestigeResult, w*sizeof(double)));

   // Launch kernel for page rank
   for (int i = 0; i < ITERATIONS; i+=2) {
      HANDLE_ERROR(hipMemcpy(dPrestigeResult, dOnes, w*sizeof(double),
                hipMemcpyDeviceToDevice));
      PageRankKernel(dVal, dRowPtr, dColInd, dPrestige, dPrestigeResult, w,
                     nnz, dOnes);
      HANDLE_ERROR(hipMemcpy(dPrestige, dOnes, w*sizeof(double),
                   hipMemcpyDeviceToDevice));
      PageRankKernel(dVal, dRowPtr, dColInd, dPrestigeResult, dPrestige, w,
                     nnz, dOnes);
   }

   // Copy result to host.
   HANDLE_ERROR(hipMemcpy(prestige, dPrestige, w*sizeof(double),
                hipMemcpyDeviceToHost));

   // Free memory.
   HANDLE_ERROR(hipFree(dOnes));
   HANDLE_ERROR(hipFree(dVal));
   HANDLE_ERROR(hipFree(dRowPtr));
   HANDLE_ERROR(hipFree(dColInd));
   HANDLE_ERROR(hipFree(dPrestige));
   HANDLE_ERROR(hipFree(dPrestigeResult));
}

void updateNodePrestige(vector<Node *> nodes, double *prestige) {
   for (int i = 0; i < nodes.size(); i++) {
      nodes[i]->updatePrestige(prestige[i]);
   }
}

void pageRank(GraphUtils::NodeGraph *graph) {
   CSRMatrix *matrix = listToCSRMatrix(graph);
   const int width = matrix->width;
   double *prestige = createPrestigeVector(width);
   double *ones = (double *) malloc(width*sizeof(double));
   for (int i = 0; i < width; i++)
      ones[i] = 1;

   PageRankOnDevice(matrix, prestige, ones);

   // update Node objects in vertex
   updateNodePrestige(matrix->nodes, prestige);
   free(prestige);

   // sort result and print out ranking
   std::sort(matrix->nodes.begin(), matrix->nodes.end(), Node::CompareByRank());
   for (int i = 0; i < matrix->nodes.size(); i++) {
      Node *node = matrix->nodes[i];
      printf("%d: %s with rank %f\n",
             i + 1, node->identifier.c_str(), node->curRank); 
   }
}

int main(int argc, char **argv)
{
   string fileName = string(argv[1]);
   GraphUtils::NodeGraph *graph;

   //Check the filename to see if its SNAP or CSV
   if(fileName.substr(fileName.find_last_of(".") + 1) == "csv") {
      graph = readCSVData(argv[1]);
   }
   else {
      graph = readSNAPData(argv[1]);
   }

   pageRank(graph);
   return 0;
}
