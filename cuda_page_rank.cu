#include "hip/hip_runtime.h"
#include "PageRank.h"
#include "GraphUtils.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <fcntl.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#define TILEWIDTH 32
#ifndef ITERATIONS
#define ITERATIONS 1000
#endif

#ifndef D
#define D 0.85
#endif

#ifndef max
#define max(a,b) ((a > b) ? a : b)
#endif

// Taken from here: http://stackoverflow.com/questions/13041399/equivalent-of-cudageterrorstring-for-cublas
/*
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}
*/

// Handles an error for cuda calls.
static void HandleError( hipError_t err,
    const char *file,
    int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
    exit( EXIT_FAILURE );
  }
}

/*
static void CheckCublasError(hipblasStatus_t err, const char *file, int line) {
   if (err != HIPBLAS_STATUS_SUCCESS) {
    printf("%s in %s at line %d\n", _cudaGetErrorEnum(err), file, line);
    exit( EXIT_FAILURE );
   }
}
*/

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
//#define HANDLE_CUBLAS_ERROR( err ) (CheckCublasError( err, __FILE__, __LINE__ ))

// Exits gracefully if malloc returns a NULL pointer.
void *check_malloc(size_t size) {
   void *ptr = malloc(size);
   if (ptr == NULL) {
      fprintf(stderr, "malloc failed.\n");
      exit(-1);
   }
   return ptr;
}

__global__ void AddRandomness(double *prestige, double *rPrestige, int vertices) {
   int ndx = blockIdx.x*blockDim.x + threadIdx.x;
   if (ndx < vertices) {
      rPrestige[ndx] = D*prestige[ndx] + (1-D)/(double)vertices;
   }
}

__global__ void MatMultKernel(double *Md, int rows_M, int cols_M,
                              double *Nd, int rows_N, int cols_N,
                              double *Pd, int rows_P, int cols_P,
                              int tiles) {
   __shared__ double Mds[TILEWIDTH][TILEWIDTH];
   __shared__ double Nds[TILEWIDTH][TILEWIDTH];
   int row = blockIdx.y*TILEWIDTH + threadIdx.y;
   int col = blockIdx.x*TILEWIDTH + threadIdx.x;
   double pVal = 0; 
   int m, k;
   for (m = 0; m < tiles; m++) {
      Mds[threadIdx.y][threadIdx.x] = 
         (row >= rows_M) || ((m*TILEWIDTH + threadIdx.x) >= cols_M) ?
            0 : Md[row*cols_M + (m*TILEWIDTH + threadIdx.x)];
      
      Nds[threadIdx.y][threadIdx.x] =
         (col >= cols_N) || ((m*TILEWIDTH + threadIdx.y) >= rows_N) ?
         0 : Nd[col + (m*TILEWIDTH + threadIdx.y)*cols_N];
      __syncthreads();
      for (k = 0; k < TILEWIDTH; k++) {
         pVal += Mds[threadIdx.y][k]*Nds[k][threadIdx.x];
      }
      __syncthreads();
   }
   if (row < rows_P && col < cols_P) {
      Pd[row*cols_P + col] = pVal;
   }
}

void PageRankOnDevice(double *matrix, int rows_matrix, int cols_matrix,
                      double *N, int rows_N, int cols_N) {
   double *matrixd, *Nd, *Pd;
   // Allocate memory.
   int size_matrix = rows_matrix*cols_matrix*sizeof(double);
   HANDLE_ERROR(hipMalloc(&matrixd, size_matrix));
   HANDLE_ERROR(hipMemcpy(matrixd, matrix, size_matrix, hipMemcpyHostToDevice));
   int size_N = rows_N*cols_N*sizeof(double);
   HANDLE_ERROR(hipMalloc(&Nd, size_N));
   HANDLE_ERROR(hipMemcpy(Nd, N, size_N, hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMalloc(&Pd, size_N));

   // Launch kernel for the matrix multiply.
   /*
   hipblasHandle_t handle;
   HANDLE_CUBLAS_ERROR(hipblasCreate(&handle));
   const double alpha = 1.0f;
   const double beta = 0.0f;
   HANDLE_CUBLAS_ERROR(hipblasDgemm(handle,
                       HIPBLAS_OP_N, HIPBLAS_OP_N,
                       rows_matrix, cols_N, cols_matrix,
                       &alpha,
                       matrixd, rows_matrix,
                       Nd, rows_N,
                       &beta,
                       Pd, rows_N));
*/
   int result_width = ceil(cols_N/(float)TILEWIDTH);
   int result_len = ceil(rows_N/(float)TILEWIDTH);
   dim3 dimGrid(result_width, result_len);
   dim3 dimBlock(TILEWIDTH, TILEWIDTH);
   int tiles = max(max(rows_matrix, cols_matrix), max(rows_N, cols_N));
   MatMultKernel<<<dimGrid, dimBlock>>>(matrixd, rows_matrix, cols_matrix,
                                        Nd, rows_N, cols_N,
                                        Pd, rows_N, cols_N,
                                        tiles);

   // Launch kernel for adding randomness.
   dim3 r_grid(ceil(rows_N/TILEWIDTH), 1);
   dim3 r_block(TILEWIDTH, 1);
   AddRandomness<<<r_grid, r_block>>>(Pd, Nd, rows_matrix);

   // Copy result to host.
   HANDLE_ERROR(hipMemcpy(N, Nd, size_N, hipMemcpyDeviceToHost));

   // Free memory.
   HANDLE_ERROR(hipFree(matrixd));
   HANDLE_ERROR(hipFree(Nd));
   HANDLE_ERROR(hipFree(Pd));
}

void updateNodePrestige(vector<Node *> nodes, double *prestige) {
   for (int i = 0; i < nodes.size(); i++) {
      nodes[i]->updatePrestige(prestige[i]);
   }
}

void pageRank(GraphUtils::NodeGraph *graph) {
   GraphUtils::NodeMatrix *matrix = GraphUtils::listToMatrix(graph);
   const int width = matrix->width;
   double *prestige = GraphUtils::matrixToPrestige(matrix);

   // while not converge
   for (int i = 0; i < ITERATIONS; i++) {
      PageRankOnDevice(matrix->matrix, width, width,
                                  prestige, width, 1);
   }

   // update Node objects in vertex
   updateNodePrestige(matrix->nodes, prestige);
   free(prestige);

   // sort result and print out ranking
   std::sort(matrix->nodes.begin(), matrix->nodes.end(), Node::CompareByRank());
   for (int i = 0; i < matrix->nodes.size(); i++) {
      Node *node = matrix->nodes[i];
      printf("%d: %s with rank %lf\n",
             i + 1, node->identifier.c_str(), node->curRank); 
   }
}
