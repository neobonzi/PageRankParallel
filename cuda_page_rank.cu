#include "hip/hip_runtime.h"
#include "PageRank.h"
//#include "GraphUtils.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <fcntl.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#define TILEWIDTH 32.0

#ifndef D
#define D 0.75
#endif

#ifndef max
#define max(a,b) ((a > b) ? a : b)
#endif

// Taken from here: http://stackoverflow.com/questions/13041399/equivalent-of-cudageterrorstring-for-cublas
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

// Handles an error for cuda calls.
static void HandleError( hipError_t err,
    const char *file,
    int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
    exit( EXIT_FAILURE );
  }
}

static void CheckCublasError(hipblasStatus_t err, const char *file, int line) {
   if (err != HIPBLAS_STATUS_SUCCESS) {
    printf("%s in %s at line %d\n", _cudaGetErrorEnum(err), file, line);
    exit( EXIT_FAILURE );
   }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#define HANDLE_CUBLAS_ERROR( err ) (CheckCublasError( err, __FILE__, __LINE__ ))

// Exits gracefully if malloc returns a NULL pointer.
void *check_malloc(size_t size) {
   void *ptr = malloc(size);
   if (ptr == NULL) {
      fprintf(stderr, "malloc failed.\n");
      exit(-1);
   }
   return ptr;
}

__global__ void AddRandomness(double *prestige, int vertices) {
   int ndx = blockIdx.x*blockDim.x + threadIdx.x;
   if (ndx < vertices)
      prestige[ndx] = D*prestige[ndx] + (1-D)/(double)vertices;
}
__global__ void UpdateMatrix(double *matrix, double *prestige, int width) {
   int row = blockIdx.y*TILEWIDTH + threadIdx.y;
   int col = blockIdx.x*TILEWIDTH + threadIdx.x;
   if (row < width && col < width) {
      matrix[INDEX(row, col, width)] = prestige[row];
   }
}

void PageRankOnDevice(double *matrix, int rows_matrix, int cols_matrix,
                      double *N, int rows_N, int cols_N) {
   double *matrixd, *Nd, *Pd;
   // Allocate memory.
   int size_matrix = rows_matrix*cols_matrix*sizeof(double);
   HANDLE_ERROR(hipMalloc(&matrixd, size_matrix));
   HANDLE_ERROR(hipMemcpy(matrixd, matrix, size_matrix, hipMemcpyHostToDevice));
   int size_N = rows_N*cols_N*sizeof(double);
   HANDLE_ERROR(hipMalloc(&Nd, size_N));
   HANDLE_ERROR(hipMemcpy(Nd, N, size_N, hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMalloc(&Pd, size_N));

   // Launch kernel for the matrix multiply.
   hipblasHandle_t handle;
   HANDLE_CUBLAS_ERROR(hipblasCreate(&handle));
   const double alpha = 1.0f;
   const double beta = 0.0f;
   HANDLE_CUBLAS_ERROR(hipblasDgemm(handle,
                       HIPBLAS_OP_N, HIPBLAS_OP_N,
                       rows_matrix, cols_N, cols_matrix,
                       &alpha,
                       matrixd, rows_matrix,
                       Nd, rows_N,
                       &beta,
                       Pd, rows_N));

   // Launch kernel for adding randomness.
   dim3 r_grid(ceil(rows_N/TILEWIDTH), 1);
   dim3 r_block(TILEWIDTH, 1);
   AddRandomness<<<r_grid, r_block>>>(Pd, rows_N);

   // Launch kernel to update matrix
   int grid_width = ceil(cols_matrix/(double)TILEWIDTH);
   int grid_len = ceil(rows_matrix/(double)TILEWIDTH);
   dim3 u_grid(grid_width, grid_len);
   dim3 u_block(TILEWIDTH, TILEWIDTH);
   UpdateMatrix<<<u_grid, u_block>>>(matrixd, Pd, rows_N);

   // Copy result to host.
   HANDLE_ERROR(hipMemcpy(matrix, matrixd, size_matrix, hipMemcpyDeviceToHost));
   HANDLE_ERROR(hipMemcpy(N, Pd, size_N, hipMemcpyDeviceToHost));

   // Free memory.
   HANDLE_ERROR(hipFree(matrixd));
   HANDLE_ERROR(hipFree(Nd));
   HANDLE_ERROR(hipFree(Pd));
}

void updateNodePrestige(vector<Node *> nodes, double *prestige) {
   for (int i = 0; i < nodes.size(); i++) {
      nodes[i]->updatePrestige(prestige[i]);
   }
}

void pageRank(GraphUtils::NodeGraph *graph) {
   GraphUtils::NodeMatrix *matrix = GraphUtils::listToMatrix(graph);
   const int width = matrix->width;
   double *prestige = GraphUtils::matrixToPrestige(matrix);
   
   const int iterations = 6;
   //bool converge = false; // TODO: get it working with converge

   // while not converge
   for (int i = 0; i < iterations; i++) {
      PageRankOnDevice(matrix->matrix, width, width,
                       prestige, width, 1);
      // update converge
   }

   // update Node objects in vertex
   updateNodePrestige(matrix->nodes, prestige);
   free(prestige);

   // sort result and print out ranking
   std::sort(matrix->nodes.begin(), matrix->nodes.end(), Node::CompareByRank());
   for (int i = 0; i < matrix->nodes.size(); i++) {
      Node *node = matrix->nodes[i];
      printf("%d: %s with rank %f\n",
             i, node->identifier.c_str(), node->curRank); 
   }
}
